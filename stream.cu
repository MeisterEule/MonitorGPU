#include "hip/hip_runtime.h"
# include <stdio.h>
# include <unistd.h>
# include <math.h>
# include <float.h>
# include <limits.h>
# include <sys/time.h>

#include <hip/hip_runtime.h>

#include "common.h"
#include "stream.h"

#define CUDA_CALL_SAFE(cuda_func)                                 \
do {                                                         \
   hipError_t err = cuda_func;                              \
   if (err != hipSuccess) {                                 \
      printf ("Cuda Error: %s\n", hipGetErrorString(err));  \
      exit(err);                                             \
   }                                                         \
} while (0)                                                  \

#define CUDA_CALL_UNSAFE(cuda_func) cuda_func

#ifdef _SAFE
#define CUDA_CALL CUDA_CALL_SAFE
#else
#define CUDA_CALL CUDA_CALL_UNSAFE
#endif

#define STREAM_ARRAY_SIZE_DEFAULT 10000000
#define NREPS_DEFAULT 10
 

//#ifdef NTIMES
//#if NTIMES<=1
//#   define NTIMES	10
//#endif
//#endif

#ifndef NTIMES
#   define NTIMES	10
#endif

# define HLINE "-------------------------------------------------------------\n"

# ifndef MIN
# define MIN(x,y) ((x)<(y)?(x):(y))
# endif
# ifndef MAX
# define MAX(x,y) ((x)>(y)?(x):(y))
# endif

#ifndef STREAM_TYPE
#define STREAM_TYPE double
#endif

#define SCALE_SCALAR 3.0
#define TRIAD_SCALAR 3.0

// *************************
// **** The CUDA kernels ***
// *************************

__global__ void vector_copy (STREAM_TYPE *out, STREAM_TYPE *in, long long n) {
   long long tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < n) out[tid] = in[tid];
}

__global__ void vector_scale (STREAM_TYPE *out, STREAM_TYPE *in,  long long n) {
   long long tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < n) out[tid] = SCALE_SCALAR * in[tid];
}

__global__ void vector_add (STREAM_TYPE *out, STREAM_TYPE *a, STREAM_TYPE *b, long long n) {
   long long tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < n) out[tid] = a[tid] + b[tid];
}

__global__ void vector_triad (STREAM_TYPE *out, STREAM_TYPE *b, STREAM_TYPE *c, long long n) {
   long long tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < n) out[tid] = b[tid] + TRIAD_SCALAR * c[tid];
}

static int rt_nreps;
static double rt_bytes[4] = {0.0};
static int rt_block_size;

void print_help () {
  printf ("Stream GPU options: \n");
  printf ("-s: Nr. of elements\n");
  printf ("-n: Nr. of repetitions\n");
}

//void print_header () {
//   printf(HLINE);
//   printf("STREAM CUDA $\n");
//   printf(HLINE);
//   printf ("Arrays have %d elements with %d bytes each\n", rt_array_size, sizeof(STREAM_TYPE));
//   printf ("   Total: %lf GiB\n", 3.0 * rt_array_size * sizeof(STREAM_TYPE) / 1024 / 1024 / 1024);
//}

STREAM_TYPE *a, *b, *c;
STREAM_TYPE *d_a, *d_b, *d_c;

bool setup_fields (int array_size) {
   printf ("Required memory: %lf BiB\n", (double)array_size * sizeof(STREAM_TYPE) / 1024 / 1024 / 1024);
   a = (STREAM_TYPE*) malloc (array_size * sizeof(STREAM_TYPE)); 
   b = (STREAM_TYPE*) malloc (array_size * sizeof(STREAM_TYPE)); 
   c = (STREAM_TYPE*) malloc (array_size * sizeof(STREAM_TYPE)); 
   for (int j = 0; j < array_size; j++) {
     a[j] = 2.0; b[j] = 2.0; c[j] = 0.0;
   }

   if ((hipMalloc ((void**)&d_a, sizeof(STREAM_TYPE) * array_size)) == hipErrorOutOfMemory) {
      printf ("Failed allocate field on device. Out of memory!\n");
      return false;
   }
   if ((hipMalloc ((void**)&d_b, sizeof(STREAM_TYPE) * array_size)) == hipErrorOutOfMemory) {
      printf ("Failed allocate field on device. Out of memory!\n");
      return false;
   }
   if ((hipMalloc ((void**)&d_c, sizeof(STREAM_TYPE) * array_size)) == hipErrorOutOfMemory) {
      printf ("Failed allocate field on device. Out of memory!\n");
      return false;
   }

   hipMemcpy (d_a, a, sizeof(STREAM_TYPE) * array_size, hipMemcpyHostToDevice);
   hipMemcpy (d_b, b, sizeof(STREAM_TYPE) * array_size, hipMemcpyHostToDevice);
   hipMemcpy (d_c, c, sizeof(STREAM_TYPE) * array_size, hipMemcpyHostToDevice);
   return true;
}

static double	avgtime[4] = {0}, maxtime[4] = {0},
		mintime[4] = {FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX};


//extern double mysecond();


void display_summary (double **times, int n_reps) {
   const char *label[4] = {"Copy:      ", "Scale:     ", "Add:       ", "Triad:     "};
   // First iteration is skipped
   for (int k = 1; k < NTIMES; k++) { 
       for (int j = 0; j < 4; j++) {
           avgtime[j] = avgtime[j] + times[j][k];
           mintime[j] = MIN(mintime[j], times[j][k]);
           maxtime[j] = MAX(maxtime[j], times[j][k]);
       }
   }
   
   printf ("Bytes per operation[GiB]: %lf %lf %lf %lf\n",
           rt_bytes[0] / 1024 / 1024 / 1024,
           rt_bytes[1] / 1024 / 1024 / 1024,
           rt_bytes[2] / 1024 / 1024 / 1024,
           rt_bytes[3] / 1024 / 1024 / 1024);
   printf("Function    Best Rate GB/s  Avg time     Min time     Max time\n");
   for (int j = 0; j < 4; j++) {
       	avgtime[j] = avgtime[j] / (double)(rt_nreps - 1);
       	printf("%s%12.1f  %11.6f  %11.6f  %11.6f\n", label[j],
               1.0E-09 * rt_bytes[j] / mintime[j],
               avgtime[j],
               mintime[j],
               maxtime[j]);
   }
   printf(HLINE);
}

#ifndef abs
#define abs(a) ((a) >= 0 ? (a) : -(a))
#endif
bool check_results (double **times, int array_size, int n_reps) {

   //hipMemcpy (a, d_a, sizeof(STREAM_TYPE) * array_size, hipMemcpyDeviceToHost);
   //hipMemcpy (b, d_b, sizeof(STREAM_TYPE) * array_size, hipMemcpyDeviceToHost);
   //hipMemcpy (c, d_c, sizeof(STREAM_TYPE) * array_size, hipMemcpyDeviceToHost);

   STREAM_TYPE aj ,bj, cj;
   STREAM_TYPE aSumErr,bSumErr,cSumErr;
   STREAM_TYPE aAvgErr,bAvgErr,cAvgErr;

   double epsilon;
   int	ierr,err;

    /* reproduce initialization */
	aj = 1.0;
	bj = 2.0;
	cj = 0.0;
    /* a[] is modified during timing check */
	aj = 2.0E0 * aj;
    /* now execute timing loop */
	for (int k = 0; k < n_reps; k++) {
            cj = aj;
            bj = SCALE_SCALAR * cj;
            cj = aj + bj;
            aj = bj + TRIAD_SCALAR * cj;
        }

    /* accumulate deltas between observed and expected results */
	aSumErr = 0.0;
	bSumErr = 0.0;
	cSumErr = 0.0;
	for (int j = 0; j < array_size; j++) {
           //printf ("Compare: %lf %lf\n", a[j], aj);
	   aSumErr += abs(a[j] - aj);
	   bSumErr += abs(b[j] - bj);
	   cSumErr += abs(c[j] - cj);
	}
	aAvgErr = aSumErr / (STREAM_TYPE) array_size;
	bAvgErr = bSumErr / (STREAM_TYPE) array_size;
	cAvgErr = cSumErr / (STREAM_TYPE) array_size;

	if (sizeof(STREAM_TYPE) == 4) {
           epsilon = 1.e-6;
	}
	else if (sizeof(STREAM_TYPE) == 8) {
           epsilon = 1.e-13;
	}
	else {
	   printf("WEIRD: sizeof(STREAM_TYPE) = %lu\n",sizeof(STREAM_TYPE));
	   epsilon = 1.e-6;
	}

        bool oka = abs(aAvgErr / aj) <= epsilon;
        bool okb = abs(bAvgErr / bj) <= epsilon;
        bool okc = abs(cAvgErr / cj) <= epsilon;
        printf ("Err: %lf %lf %lf\n", aAvgErr, bAvgErr, cAvgErr);
        printf ("oka: %d, okb: %d, okc: %d\n", oka, okb, okc);

        return oka && okb && okc;
}

void cleanup_pass () {
   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);
   free(a);
   free(b);
   free(c);
}

//int main(int argc, char *argv[]) {
void do_stream (int array_size, int n_times, int *status) {

    int block_size = BLOCK_SIZE_DEFAULT;
    int grid_size = (array_size + block_size) / block_size;
    double **times;

    times = (double**) malloc (4 * sizeof(double*));
    for (int i = 0; i < 4; i++) {
       times[i] = (double*) malloc (n_times * sizeof(double));
    }

    for (int k = 0; k < n_times; k++) {
       times[0][k] = 0.0;
       times[1][k] = 0.0;
       times[2][k] = 0.0;
       times[3][k] = 0.0;
    }

    double test_bytes[4];
    test_bytes[0] = 2 * sizeof(STREAM_TYPE) * array_size;
    test_bytes[1] = 2 * sizeof(STREAM_TYPE) * array_size;
    test_bytes[2] = 3 * sizeof(STREAM_TYPE) * array_size;
    test_bytes[3] = 3 * sizeof(STREAM_TYPE) * array_size;

    // Allocate host fields
    if (!(a = (STREAM_TYPE*) malloc (array_size * sizeof(STREAM_TYPE)))) {
      *status = STREAM_OOM_HOST;
      return;
    }
    if (!(b = (STREAM_TYPE*) malloc (array_size * sizeof(STREAM_TYPE)))) {
      *status = STREAM_OOM_HOST;
      return;
    } 
    if (!(c = (STREAM_TYPE*) malloc (array_size * sizeof(STREAM_TYPE)))) {
      *status = STREAM_OOM_HOST;
      return;
    }

    if ((hipMalloc ((void**)&d_a, sizeof(STREAM_TYPE) * array_size)) == hipErrorOutOfMemory) {
      *status = STREAM_OOM_DEVICE;
      return;
    }
    if ((hipMalloc ((void**)&d_b, sizeof(STREAM_TYPE) * array_size)) == hipErrorOutOfMemory) {
      *status = STREAM_OOM_DEVICE;
      return;
    }
    if ((hipMalloc ((void**)&d_c, sizeof(STREAM_TYPE) * array_size)) == hipErrorOutOfMemory) {
      *status = STREAM_OOM_DEVICE;
      return;
    }

    for (int i = 0; i < array_size; i++) {
       a[i] = 2.0;
       b[i] = 2.0;
       c[i] = 0.0; 
    }

    hipMemcpy (d_a, a, sizeof(STREAM_TYPE) * array_size, hipMemcpyHostToDevice);
    hipMemcpy (d_b, b, sizeof(STREAM_TYPE) * array_size, hipMemcpyHostToDevice);
    hipMemcpy (d_c, c, sizeof(STREAM_TYPE) * array_size, hipMemcpyHostToDevice);

    double t1, t2;
    for (int k = 0; k < n_times; k++) {
       times[0][k] = get_time_monotonic();
       vector_copy<<<grid_size,block_size>>> (d_c, d_a, array_size);
       hipDeviceSynchronize();
       times[0][k] = get_time_monotonic() - times[0][k];

       times[1][k] = get_time_monotonic();
       vector_scale<<<grid_size,block_size>>> (d_b, d_c, array_size);
       hipDeviceSynchronize();
       times[1][k] = get_time_monotonic() - times[1][k];

       times[2][k] = get_time_monotonic();
       vector_add<<<grid_size,block_size>>> (d_c, d_a, d_b, array_size);
       hipDeviceSynchronize();
       times[2][k] = get_time_monotonic() - times[2][k];

       times[3][k] = get_time_monotonic();
       vector_triad<<<grid_size,block_size>>> (d_a, d_b, d_c, array_size);
       hipDeviceSynchronize();
       times[3][k] = get_time_monotonic() - times[3][k];
    }

    hipMemcpy (a, d_a, sizeof(STREAM_TYPE) * array_size, hipMemcpyDeviceToHost);
    hipMemcpy (b, d_b, sizeof(STREAM_TYPE) * array_size, hipMemcpyDeviceToHost);
    hipMemcpy (c, d_c, sizeof(STREAM_TYPE) * array_size, hipMemcpyDeviceToHost);

    bool check = check_results (times, array_size, n_times);
    if (!check) {
       *status = STREAM_INVALID_RESULTS;
       return;
    }

    for (int i = 0; i < n_times; i++) {
       for (int j = 0; j < 4; j++) {
          avgtime[j] = avgtime[j] + times[j][i];
          if (times[j][i] < mintime[j]) mintime[j] = times[j][i];
       }
    }
    for (int i = 0; i < 4; i++) {
       printf ("Avg: %lf\n", avgtime[i] / (n_times - 1));
       printf ("Min: %lf\n", mintime[i]);
       printf ("Rate: %lf\n", 1.0e-9 * test_bytes[i] / mintime[i]);
       printf ("***************************\n");
    }
    printf(HLINE);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    *status = STREAM_SUCCESS;
}

/* A gettimeofday routine to give access to the wall
   clock timer on most UNIX-like systems.  */

//#include <sys/time.h>
//
//double mysecond()
//{
//        struct timeval tp;
//        struct timezone tzp;
//        int i;
//
//        i = gettimeofday(&tp,&tzp);
//        return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
//}


