#include "hip/hip_runtime.h"
# include <stdio.h>
# include <unistd.h>
# include <math.h>
# include <float.h>
# include <limits.h>
# include <sys/time.h>

#include <hip/hip_runtime.h>

#include "common.h"
#include "stream.h"

__global__ void vector_copy (STREAM_TYPE *out, STREAM_TYPE *in, long long n) {
   long long tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < n) out[tid] = in[tid];
}

__global__ void vector_scale (STREAM_TYPE *out, STREAM_TYPE *in,  long long n) {
   long long tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < n) out[tid] = SCALE_SCALAR * in[tid];
}

__global__ void vector_add (STREAM_TYPE *out, STREAM_TYPE *a, STREAM_TYPE *b, long long n) {
   long long tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < n) out[tid] = a[tid] + b[tid];
}

__global__ void vector_triad (STREAM_TYPE *out, STREAM_TYPE *b, STREAM_TYPE *c, long long n) {
   long long tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < n) out[tid] = b[tid] + TRIAD_SCALAR * c[tid];
}



//void display_summary (double **times, int n_reps) {
//   const char *label[4] = {"Copy:      ", "Scale:     ", "Add:       ", "Triad:     "};
//   // First iteration is skipped
//   for (int k = 1; k < NTIMES; k++) { 
//       for (int j = 0; j < 4; j++) {
//           avgtime[j] = avgtime[j] + times[j][k];
//           mintime[j] = MIN(mintime[j], times[j][k]);
//           maxtime[j] = MAX(maxtime[j], times[j][k]);
//       }
//   }
//   
//   printf ("Bytes per operation[GiB]: %lf %lf %lf %lf\n",
//           rt_bytes[0] / 1024 / 1024 / 1024,
//           rt_bytes[1] / 1024 / 1024 / 1024,
//           rt_bytes[2] / 1024 / 1024 / 1024,
//           rt_bytes[3] / 1024 / 1024 / 1024);
//   printf("Function    Best Rate GB/s  Avg time     Min time     Max time\n");
//   for (int j = 0; j < 4; j++) {
//       	avgtime[j] = avgtime[j] / (double)(rt_nreps - 1);
//       	printf("%s%12.1f  %11.6f  %11.6f  %11.6f\n", label[j],
//               1.0E-09 * rt_bytes[j] / mintime[j],
//               avgtime[j],
//               mintime[j],
//               maxtime[j]);
//   }
//   printf(HLINE);
//}

bool check_results (STREAM_TYPE *a, STREAM_TYPE *b, STREAM_TYPE *c,
                    double **times, int array_size, int n_reps) {

   STREAM_TYPE aj ,bj, cj;
   STREAM_TYPE a_sum_err, b_sum_err, c_summ_err;
   STREAM_TYPE a_avg_err, b_avg_err, c_agv_err;

   double epsilon;
   int	ierr,err;

    aj = 1.0;
    bj = 2.0;
    cj = 0.0;
    // a[] is modified during timing check
    aj = 2.0E0 * aj;
    for (int k = 0; k < n_reps; k++) {
       cj = aj;
       bj = SCALE_SCALAR * cj;
       cj = aj + bj;
       aj = bj + TRIAD_SCALAR * cj;
    }

    /* accumulate deltas between observed and expected results */
    a_sum_err = 0.0;
    b_sum_err = 0.0;
    c_summ_err = 0.0;

#ifndef abs
#define abs(a) ((a) >= 0 ? (a) : -(a))
#endif
    for (int j = 0; j < array_size; j++) {
       a_sum_err += abs(a[j] - aj);
       b_sum_err += abs(b[j] - bj);
       c_summ_err += abs(c[j] - cj);
    }
    a_avg_err = a_sum_err / (STREAM_TYPE) array_size;
    b_avg_err = b_sum_err / (STREAM_TYPE) array_size;
    c_agv_err = c_summ_err / (STREAM_TYPE) array_size;

    if (sizeof(STREAM_TYPE) == 4) {
        epsilon = 1.e-6;
    } else if (sizeof(STREAM_TYPE) == 8) {
        epsilon = 1.e-13;
    }

    bool oka = abs(a_avg_err / aj) <= epsilon;
    bool okb = abs(b_avg_err / bj) <= epsilon;
    bool okc = abs(c_agv_err / cj) <= epsilon;
    return oka && okb && okc;
}

void do_stream (int array_size, int n_times,
                double *best_copy, double *best_scale, 
                double *best_add, double *best_triad, int *status) {
 
    int block_size = BLOCK_SIZE_DEFAULT;
    int grid_size = (array_size + block_size) / block_size;

    double **times = (double**) malloc (4 * sizeof(double*));
    for (int i = 0; i < 4; i++) {
       times[i] = (double*) malloc (n_times * sizeof(double));
    }

    for (int i = 0; i < 4; i++) {
       for (int j = 0; j < n_times; j++) {
          times[i][j] = 0.0;
       }
    }

    double test_bytes[4];
    test_bytes[STREAM_COPY] = 2 * sizeof(STREAM_TYPE) * array_size;
    test_bytes[STREAM_SCALE] = 2 * sizeof(STREAM_TYPE) * array_size;
    test_bytes[STREAM_ADD] = 3 * sizeof(STREAM_TYPE) * array_size;
    test_bytes[STREAM_TRIAD] = 3 * sizeof(STREAM_TYPE) * array_size;

    // Host fields
    STREAM_TYPE *a, *b, *c;
    // Device fields
    STREAM_TYPE *d_a, *d_b, *d_c;

    // Allocate host fields
    if (!(a = (STREAM_TYPE*) malloc (array_size * sizeof(STREAM_TYPE)))) {
      *status = STREAM_OOM_HOST;
      return;
    }
    if (!(b = (STREAM_TYPE*) malloc (array_size * sizeof(STREAM_TYPE)))) {
      *status = STREAM_OOM_HOST;
      return;
    } 
    if (!(c = (STREAM_TYPE*) malloc (array_size * sizeof(STREAM_TYPE)))) {
      *status = STREAM_OOM_HOST;
      return;
    }

    if ((hipMalloc ((void**)&d_a, sizeof(STREAM_TYPE) * array_size)) == hipErrorOutOfMemory) {
      *status = STREAM_OOM_DEVICE;
      return;
    }
    if ((hipMalloc ((void**)&d_b, sizeof(STREAM_TYPE) * array_size)) == hipErrorOutOfMemory) {
      *status = STREAM_OOM_DEVICE;
      return;
    }
    if ((hipMalloc ((void**)&d_c, sizeof(STREAM_TYPE) * array_size)) == hipErrorOutOfMemory) {
      *status = STREAM_OOM_DEVICE;
      return;
    }

    for (int i = 0; i < array_size; i++) {
       a[i] = 2.0;
       b[i] = 2.0;
       c[i] = 0.0; 
    }

    hipMemcpy (d_a, a, sizeof(STREAM_TYPE) * array_size, hipMemcpyHostToDevice);
    hipMemcpy (d_b, b, sizeof(STREAM_TYPE) * array_size, hipMemcpyHostToDevice);
    hipMemcpy (d_c, c, sizeof(STREAM_TYPE) * array_size, hipMemcpyHostToDevice);

    double t1, t2;
    for (int k = 0; k < n_times; k++) {
       times[STREAM_COPY][k] = get_time_monotonic();
       vector_copy<<<grid_size,block_size>>> (d_c, d_a, array_size);
       hipDeviceSynchronize();
       times[STREAM_COPY][k] = get_time_monotonic() - times[STREAM_COPY][k];

       times[STREAM_SCALE][k] = get_time_monotonic();
       vector_scale<<<grid_size,block_size>>> (d_b, d_c, array_size);
       hipDeviceSynchronize();
       times[STREAM_SCALE][k] = get_time_monotonic() - times[STREAM_SCALE][k];

       times[STREAM_ADD][k] = get_time_monotonic();
       vector_add<<<grid_size,block_size>>> (d_c, d_a, d_b, array_size);
       hipDeviceSynchronize();
       times[STREAM_ADD][k] = get_time_monotonic() - times[STREAM_ADD][k];

       times[STREAM_TRIAD][k] = get_time_monotonic();
       vector_triad<<<grid_size,block_size>>> (d_a, d_b, d_c, array_size);
       hipDeviceSynchronize();
       times[STREAM_TRIAD][k] = get_time_monotonic() - times[STREAM_TRIAD][k];
    }

    hipMemcpy (a, d_a, sizeof(STREAM_TYPE) * array_size, hipMemcpyDeviceToHost);
    hipMemcpy (b, d_b, sizeof(STREAM_TYPE) * array_size, hipMemcpyDeviceToHost);
    hipMemcpy (c, d_c, sizeof(STREAM_TYPE) * array_size, hipMemcpyDeviceToHost);

    bool check = check_results (a, b, c, times, array_size, n_times);
    if (!check) {
       *status = STREAM_INVALID_RESULTS;
       printf ("Fields not OK\n");
       return;
    }

    double avgtime[4] = {0};
    double maxtime[4] = {0};
    double mintime[4] = {FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX};

    for (int i = 0; i < n_times; i++) {
       for (int j = 0; j < 4; j++) {
          avgtime[j] = avgtime[j] + times[j][i];
          if (times[j][i] < mintime[j]) mintime[j] = times[j][i];
          if (times[j][i] > maxtime[j]) maxtime[j] = times[j][i];
       }
    }

    *best_copy = test_bytes[STREAM_COPY] / mintime[STREAM_COPY] * 1e-9;
    *best_scale = test_bytes[STREAM_SCALE] / mintime[STREAM_SCALE] * 1e-9;
    *best_add = test_bytes[STREAM_ADD] / mintime[STREAM_ADD] * 1e-9;
    *best_triad = test_bytes[STREAM_TRIAD] / mintime[STREAM_TRIAD] * 1e-9;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    *status = STREAM_SUCCESS;
}

